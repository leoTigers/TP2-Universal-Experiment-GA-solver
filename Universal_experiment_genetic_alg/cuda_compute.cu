#include "hip/hip_runtime.h"
#include "cuda_compute.cuh"

void repr(Grid& individual);


__global__ void cuda_eval(Parameters* d_params, Grid* population, int* scores) {
    
    int indice = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (indice >= d_params->population_size)
        return;
    //if grid hasen't changed
    /*if (!population[indice].changed)
    {
        scores[indice] = population[indice].score;
        return;
    }*/


    char movements[8][2] = { {-1, -1}, {0, -1}, {1, -1}, {1, 0}, {1, 1}, {0, 1}, {-1, 1}, {-1, 0} };
    // starting pos
    char x = -1, y = 6;
    char cur_dir = 3;

    bool end = false;
    int score = 0;
    char uses;
    char type;
    char dir;
    int cc = 0;

    //copy_tab(individual, population[indice]);
    for (int i = 0; i < DIM; ++i){
        for (int j = 0; j < DIM; ++j) {
            population[indice].cases[i][j].current_uses = 0;
            population[indice].cases[i][j].c_dir = population[indice].cases[i][j].dir;
        }
    }

    do {
        cc++;
        if (cc > 50000) {
            scores[indice] = 50000;
            return;
        }
        //repr(population[indice]);
        //move
        x += movements[cur_dir][0];
        y += movements[cur_dir][1];

        //check exit cond
        if (x < 0 || x >= DIM || y < 0 || y >= DIM) {
            end = true;
        }
        else {
            switch (population[indice].cases[x][y].type) {
            case Object_type::ARROW:
                type = population[indice].cases[x][y].o_type;
                uses = population[indice].cases[x][y].current_uses;
                dir = population[indice].cases[x][y].c_dir;
                if ((type == Arrow_type::ONE_USE && uses < 1) || (type == Arrow_type::THREE_USES && uses < 3) || (type == Arrow_type::FIVE_USES && uses < 5)) {
                    population[indice].cases[x][y].current_uses++;
                    cur_dir = dir;
                }
                else if (type == Arrow_type::INFINITE_USES) {
                    cur_dir = dir;
                }
                else if (type == Arrow_type::ROTATING) {
                    cur_dir = dir;
                    population[indice].cases[x][y].c_dir = (dir + 1) & 7;
                }
                break;
            case Object_type::ORB:
                switch (population[indice].cases[x][y].o_type)
                {
                case Orb_type::NORMAL:
                    score--;
                    break;
                case Orb_type::REFRESH:
                    if (population[indice].cases[x][y].current_uses == 0) { //refresh not used
                        for (int i = 0; i < DIM; ++i) {
                            for (int j = 0; j < DIM; ++j) {
                                population[indice].cases[i][j].current_uses = 0;
                                //population[indice].cases[i][j].c_dir = population[indice].cases[i][j].dir;
                            }
                        }
                        population[indice].cases[x][y].current_uses = 1;
                    }
                    break;
                }
                break;
            case Object_type::REFLECT: //reflect
                uses = population[indice].cases[x][y].current_uses;
                if (uses < 2) {
                    uses++;
                    if (cur_dir & 1)
                        cur_dir = (cur_dir + 4) % 8;
                    else if (cur_dir == 0)
                        cur_dir = 6;
                    else
                        cur_dir -= 2;

                    population[indice].cases[x][y].current_uses++;
                }
                break;
            }
        }
    } while (!end);
    population[indice].score = score;
    population[indice].changed = false;
    scores[indice] = score;
    //printf("%d\t%d\n", indice, score);
}

__global__ void cuda_breed(Parameters *d_params, Grid* d_population, int* d_scores, 
    int* d_scores_indices, hiprandState* state) {

    int indice = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (indice >= d_params->population_size)
        return;

    if (indice < d_params->retain_rate * d_params->population_size)
        return;

    hiprandState localState = state[indice];

    int parent_a, parent_b;
    int smod = d_params->retain_rate * d_params->population_size;
    parent_a = d_scores_indices[hiprand(&localState) % smod];
    parent_b = d_scores_indices[hiprand(&localState) % smod];

    // Reset limits for new individual
    for (int i = 0; i < 4; i++)
        d_population[d_scores_indices[indice]].limits[i] = 0;

    int crosspoint = hiprand(&localState) % 50; // the case at which point we'll take values from parent_b instead of parent_a
    for (int i = 0; i < DIM; ++i) {
        for (int j = 0; j < DIM; ++j) {
            Case tmp;
            if (i * DIM + j < crosspoint) {
                // Normal addition by crosspoint
                tmp = d_population[parent_a].cases[i][j];
                switch (tmp.type) {
                case Object_type::ARROW:
                    if (tmp.o_type > 1)
                        d_population[d_scores_indices[indice]].limits[tmp.o_type - 2]++;
                    break;
                case Object_type::ORB:
                    if (tmp.o_type == Orb_type::REFRESH)
                        d_population[d_scores_indices[indice]].limits[3]++;
                    break;
                }
            }
            else { // Addition with limit check
                tmp = d_population[parent_b].cases[i][j];
                int valid_insertion = true;
                switch (tmp.type) {
                case Object_type::ARROW:
                    switch (tmp.o_type)
                    {
                    case Arrow_type::FIVE_USES:
                        if (d_population[d_scores_indices[indice]].limits[0] >= MAX_5T_ARROWS)
                            valid_insertion = false;
                        break;
                    case Arrow_type::INFINITE_USES:
                        if (d_population[d_scores_indices[indice]].limits[1] >= MAX_INF_ARROWS)
                            valid_insertion = false;
                        break;
                    case Arrow_type::ROTATING:
                        if (d_population[d_scores_indices[indice]].limits[2] >= MAX_ROT_ARROWS)
                            valid_insertion = false;
                        break;
                    default:
                        break;
                    }
                    if (!valid_insertion) {
                        char type = hiprand(&localState) % (REFLECT_UNLOCK ? 3 : 2);
                        tmp.type = type;
                        switch (type) {
                        case Object_type::ARROW: //arrow
                            // pick an arrow type within grid object limits
                            do {
                                tmp.o_type = hiprand(&localState) % 5;
                            } while ((tmp.o_type == Arrow_type::FIVE_USES && d_population[d_scores_indices[indice]].limits[0] >= MAX_5T_ARROWS) ||
                                (tmp.o_type == Arrow_type::INFINITE_USES && d_population[d_scores_indices[indice]].limits[1] >= MAX_INF_ARROWS) ||
                                (tmp.o_type == Arrow_type::ROTATING && d_population[d_scores_indices[indice]].limits[2] >= MAX_ROT_ARROWS));

                            if (tmp.o_type > 1)
                                d_population[d_scores_indices[indice]].limits[tmp.o_type - 2]++;
                            tmp.dir = hiprand(&localState) & 7;
                            break;
                        case Object_type::ORB: //orb
                            // orb can be normal or refresh
                            tmp.o_type = (d_population[d_scores_indices[indice]].limits[3] < MAX_REFRESH) ? hiprand(&localState) & 1 : 0;
                            d_population[d_scores_indices[indice]].limits[3] += (tmp.o_type & 1);
                            break;
                        }
                    }
                    else {
                        if (tmp.o_type > 1)
                            d_population[d_scores_indices[indice]].limits[tmp.o_type - 2]++;
                    }
                    break;
                case Object_type::ORB:
                    if (tmp.o_type == Orb_type::REFRESH) {
                        if (d_population[d_scores_indices[indice]].limits[3] >= MAX_REFRESH)
                        {
                            char type = hiprand(&localState) % (REFLECT_UNLOCK ? 3 : 2);
                            tmp.type = type;
                            switch (type) {
                            case Object_type::ARROW: //arrow
                                // pick an arrow type within grid object limits
                                do {
                                    tmp.o_type = hiprand(&localState) % 5;
                                } while ((tmp.o_type == Arrow_type::FIVE_USES && d_population[d_scores_indices[indice]].limits[0] == MAX_5T_ARROWS) ||
                                    (tmp.o_type == Arrow_type::INFINITE_USES && d_population[d_scores_indices[indice]].limits[1] == MAX_INF_ARROWS) ||
                                    (tmp.o_type == Arrow_type::ROTATING && d_population[d_scores_indices[indice]].limits[2] == MAX_ROT_ARROWS));

                                if (tmp.o_type > 1)
                                    d_population[d_scores_indices[indice]].limits[tmp.o_type - 2]++;
                                tmp.dir = hiprand(&localState) & 7;
                                break;
                            case Object_type::ORB: //orb
                                // orb can be normal or refresh
                                tmp.o_type = (d_population[d_scores_indices[indice]].limits[3] < MAX_REFRESH) ? hiprand(&localState) & 1 : 0;
                                d_population[d_scores_indices[indice]].limits[3] += (tmp.o_type & 1);
                                break;
                            }
                        }
                        else {
                            d_population[d_scores_indices[indice]].limits[3]++;
                        }
                    }
                    break;
                }
            }
            d_population[d_scores_indices[indice]].cases[i][j] = tmp;
        }
    }
    d_population[d_scores_indices[indice]].changed = true;
    state[indice] = localState;
}


__global__ void cuda_mutate(Parameters* d_params, Grid* d_population, hiprandState* state) {
    int indice = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (indice >= d_params->population_size || indice == 0)
        return;

    hiprandState localState = state[indice];

    for (int i = 0; i < d_params->min_mutations; ++i) {
        // pick a random position
        int x = hiprand(&localState) % DIM;
        int y = hiprand(&localState) % DIM;

        int type;

        // update limits
        switch (d_population[indice].cases[x][y].type) {
        case Object_type::ARROW:
            type = d_population[indice].cases[x][y].o_type;
            switch (type) {
            case Arrow_type::FIVE_USES:
                d_population[indice].limits[0]--;
                break;
            case Arrow_type::INFINITE_USES:
                d_population[indice].limits[1]--;
                break;
            case Arrow_type::ROTATING:
                d_population[indice].limits[2]--;
                break;
            }
            break;
        case Object_type::ORB:
            if (d_population[indice].cases[x][y].o_type == Orb_type::REFRESH)
                d_population[indice].limits[3]--;
            break;
        }

        // create new object
        type = hiprand(&localState) % (REFLECT_UNLOCK ? 3 : 2);
        d_population[indice].cases[x][y].type = type;
        switch (type) {
        case Object_type::ARROW: //arrow
            // pick an arrow type within grid object limits
            do {
                d_population[indice].cases[x][y].o_type = hiprand(&localState) % 5;
            } while ((d_population[indice].cases[x][y].o_type == Arrow_type::FIVE_USES && d_population[indice].limits[0] == MAX_5T_ARROWS) ||
                (d_population[indice].cases[x][y].o_type == Arrow_type::INFINITE_USES && d_population[indice].limits[1] == MAX_INF_ARROWS) ||
                (d_population[indice].cases[x][y].o_type == Arrow_type::ROTATING && d_population[indice].limits[2] == MAX_ROT_ARROWS));

            if (d_population[indice].cases[x][y].o_type > 1)
                d_population[indice].limits[d_population[indice].cases[x][y].o_type - 2]++;
            d_population[indice].cases[x][y].dir = hiprand(&localState) & 7;
            break;
        case Object_type::ORB: //orb
            // orb can be normal or refresh
            d_population[indice].cases[x][y].o_type = (d_population[indice].limits[3] < MAX_REFRESH) ? hiprand(&localState) & 1 : 0;
            d_population[indice].limits[3] += (d_population[indice].cases[x][y].o_type & 1);
            break;
        }
        d_population[indice].changed = true;
    }
    
    for (int i = 0; i < d_params->max_mutations-d_params->min_mutations; ++i) {
        if (hiprand_uniform(&localState) < d_params->mutation_rate) {
            // pick a random position
            int x = hiprand(&localState) % DIM;
            int y = hiprand(&localState) % DIM;
           
            int type;

            // update limits
            switch (d_population[indice].cases[x][y].type) {
            case Object_type::ARROW:
                type = d_population[indice].cases[x][y].o_type;
                switch (type) {
                case Arrow_type::FIVE_USES:
                    d_population[indice].limits[0]--;
                    break;
                case Arrow_type::INFINITE_USES:
                    d_population[indice].limits[1]--;
                    break;
                case Arrow_type::ROTATING:
                    d_population[indice].limits[2]--;
                    break;
                }
                break;
            case Object_type::ORB:
                if (d_population[indice].cases[x][y].o_type == Orb_type::REFRESH)
                    d_population[indice].limits[3]--;
                break;
            }

            // create new object
            type = hiprand(&localState) % (REFLECT_UNLOCK ? 3 : 2);
            d_population[indice].cases[x][y].type = type;
            switch (type) {
            case Object_type::ARROW: //arrow
                // pick an arrow type within grid object limits
                do {
                    d_population[indice].cases[x][y].o_type = hiprand(&localState) % 5;
                } while ((d_population[indice].cases[x][y].o_type == Arrow_type::FIVE_USES && d_population[indice].limits[0] == MAX_5T_ARROWS) ||
                    (d_population[indice].cases[x][y].o_type == Arrow_type::INFINITE_USES && d_population[indice].limits[1] == MAX_INF_ARROWS) ||
                    (d_population[indice].cases[x][y].o_type == Arrow_type::ROTATING && d_population[indice].limits[2] == MAX_ROT_ARROWS));

                if (d_population[indice].cases[x][y].o_type > 1)
                    d_population[indice].limits[d_population[indice].cases[x][y].o_type - 2]++;
                d_population[indice].cases[x][y].dir = hiprand(&localState) & 7;
                break;
            case Object_type::ORB: //orb
                // orb can be normal or refresh
                d_population[indice].cases[x][y].o_type = (d_population[indice].limits[3] < MAX_REFRESH) ? hiprand(&localState) & 1 : 0;
                d_population[indice].limits[3] += (d_population[indice].cases[x][y].o_type & 1);
                break;
            }
            d_population[indice].changed = true;
        }
    }
    state[indice] = localState;
}

void quicksortIndices(int population_size, int values[], int indices[]) {
    for (int i = 0; i < population_size; i++)
        indices[i] = i;
    quicksortIndices(population_size, values, indices, 0, population_size - 1);
}

/**
 * @brief Sorts the provided values between two indices while applying the same
 *        transformations to the array of indices
 *
 * @param values  the values to sort
 * @param indices the indices to sort according to the corresponding values
 * @param         low, high are the **inclusive** bounds of the portion of array
 *                to sort
 */
void quicksortIndices(int population_size, int values[], int indices[], int low, int high) {
    int l = low;
    int h = high;
    int pivot = values[l];
    while (l <= h) {
        if (values[l] < pivot)
            l++;
        else if (values[h] > pivot)
            h--;
        else {
            swap(l, h, values, indices);
            l++;
            h--;
        }
    }
    if (low < h)
        quicksortIndices(population_size, values, indices, low, h);
    if (high > l)
        quicksortIndices(population_size, values, indices, l, high);
}

/**
 * @brief Swaps the elements of the given arrays at the provided positions
 *
 * @param         i, j the indices of the elements to swap
 * @param values  the array floats whose values are to be swapped
 * @param indices the array of ints whose values are to be swapped
 */
void swap(int i, int j, int values[], int indices[]) {
    int tempValue = values[i];
    int tempIndice = indices[i];
    values[i] = values[j];
    indices[i] = indices[j];
    values[j] = tempValue;
    indices[j] = tempIndice;
}

__global__ void setup_kernel(hiprandState* state, int* d_val) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(d_val[id], id, 0, &state[id]);
}

void setup(hiprandState* state, Parameters *params) {
    int  *values, *d_values;
    values = new int[(params->population_size / THREADS + 1)];
    hipMalloc((void**)&d_values, sizeof(int) * (params->population_size / THREADS + 1));
    for (int i = 0; i < (params->population_size / THREADS + 1); ++i)
        values[i] = rand();
    hipMemcpy(d_values, values, sizeof(int) * (params->population_size / THREADS + 1), hipMemcpyHostToDevice);
    setup_kernel<<<(params->population_size / THREADS + 1), THREADS>>>(state, d_values);
    hipFree(d_values);
    delete values;
}

void cuda_run(Parameters params, Parameters* d_params, Grid* population, Grid* d_population, 
    int* scores, int *d_scores, int* scores_indices, int *d_scores_indices, 
    Grid *fittest, Grid *d_fittest, hiprandState* state) {
    
    // evaluate the population
    cuda_eval<<<(params.population_size / THREADS + 1), THREADS>>>(d_params, d_population, d_scores);
    hipDeviceSynchronize();

    // get back the scores
    hipMemcpy(scores, d_scores, sizeof(int) * params.population_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < params.population_size; ++i) {
        scores_indices[i] = i;
        if (scores[i] == 50000) {
            hipMemcpy(fittest, &d_population[i], sizeof(Grid), hipMemcpyDeviceToHost);
            std::cout << "ERROR" << std::endl;
            repr(*fittest);
        }
    }
    quicksortIndices(params.population_size, scores, scores_indices);
    
    //save the fittest and his score
    if (scores[0] < fittest->score) {
        hipMemcpy(fittest, &d_population[scores_indices[0]], sizeof(Grid), hipMemcpyDeviceToHost);
    }

    hipMemcpy(d_scores, scores, sizeof(int) * params.population_size, hipMemcpyHostToDevice);
    hipMemcpy(d_scores_indices, scores_indices, sizeof(int) * params.population_size, hipMemcpyHostToDevice);

    // breed new elements
    cuda_breed<<<(params.population_size / THREADS + 1), THREADS>>>(d_params, d_population, d_scores, d_scores_indices, state);
    hipDeviceSynchronize(); 

    // mutate some elements
    cuda_mutate<<<(params.population_size / THREADS + 1), THREADS>>>(d_params, d_population, state);
    hipError_t a = hipDeviceSynchronize();
    //hipMemcpy(population, d_population, sizeof(Grid) * params.population_size, hipMemcpyDeviceToHost);

    //hipFree(d_population);
    //hipFree(d_population_size);
}

